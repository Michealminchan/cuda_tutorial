#include "hip/hip_runtime.h"
#include <wb.h>

#define NUM_BINS 128

#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

__global__ void histogram_kernel(const char *input, unsigned int *bins,
                                 unsigned int num_elements,
                                 unsigned int num_bins) {

  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

  // Privatized bins
  extern __shared__ unsigned int bins_s[];
  for (unsigned int binIdx = threadIdx.x; binIdx < num_bins;
       binIdx += blockDim.x) {
    bins_s[binIdx] = 0;
  }
  __syncthreads();

  // Histogram
  for (unsigned int i = tid; i < num_elements;
       i += blockDim.x * gridDim.x) {
    atomicAdd(&(bins_s[(unsigned int)input[i]]), 1);
  }
  __syncthreads();

  // Commit to global memory
  for (unsigned int binIdx = threadIdx.x; binIdx < num_bins;
       binIdx += blockDim.x) {
    atomicAdd(&(bins[binIdx]), bins_s[binIdx]);
  }
}

void histogram(const char *input, unsigned int *bins,
               unsigned int num_elements, unsigned int num_bins) {

  // zero out bins
  CUDA_CHECK(hipMemset(bins, 0, num_bins * sizeof(unsigned int)));
  // Launch histogram kernel on the bins
  {
    dim3 blockDim(256), gridDim(30);
    histogram_kernel<<<gridDim, blockDim,
                       num_bins * sizeof(unsigned int)>>>(
        input, bins, num_elements, num_bins);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
  }
}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int inputLength;
  char *hostInput;
  unsigned int *hostBins;
  char *deviceInput;
  unsigned int *deviceBins;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput =
      (char *)wbImport(wbArg_getInputFile(args, 0), &inputLength, "Text");
  hostBins = (unsigned int *)malloc(NUM_BINS * sizeof(unsigned int));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);
  wbLog(TRACE, "The number of bins is ", NUM_BINS);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  CUDA_CHECK(hipMalloc((void **)&deviceInput, inputLength));
  CUDA_CHECK(
      hipMalloc((void **)&deviceBins, NUM_BINS * sizeof(unsigned int)));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  CUDA_CHECK(hipMemcpy(deviceInput, hostInput, inputLength,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  // Launch kernel
  // ----------------------------------------------------------
  wbLog(TRACE, "Launching kernel");
  wbTime_start(Compute, "Performing CUDA computation");
  // @@ Insert code here
  histogram(deviceInput, deviceBins, inputLength, NUM_BINS);
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  CUDA_CHECK(hipMemcpy(hostBins, deviceBins,
                        NUM_BINS * sizeof(unsigned int),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  CUDA_CHECK(hipFree(deviceInput));
  CUDA_CHECK(hipFree(deviceBins));
  wbTime_stop(GPU, "Freeing GPU Memory");

  // Verify correctness
  // -----------------------------------------------------
  wbSolution(args, hostBins, NUM_BINS);

  free(hostBins);
  free(hostInput);
  return 0;
}
