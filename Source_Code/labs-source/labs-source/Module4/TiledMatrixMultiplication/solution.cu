#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)


#define TILE_WIDTH 16

// Compute C = A * B
__global__ void matrixMultiplyShared(float *A, float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns) {
  //@@ Insert code to implement matrix multiplication here
  //@@ You have to use shared memory for this lab
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

    int tx = threadIdx.x; 
    int ty = threadIdx.y; 
    
   float Cvalue = 0;
   //Loop over the A and B tiles required to compute the P element
   for(int p = 0; p < ceil(((float)numAColumns)/TILE_WIDTH) ; p++)
   {
       // Collaborative loading of A and B tiles into shared memory
       if((row < numARows)&&(p*TILE_WIDTH + threadIdx.x < numAColumns))
       {
          ds_A[ty][tx] = A[row * numAColumns + p * TILE_WIDTH + threadIdx.x];
       }else
       {
           ds_A[ty][tx] = 0;
       }
       if((p*TILE_WIDTH + threadIdx.y < numBRows)&&(col < numBColumns))
       {
         ds_B[ty][tx] = B[(p * TILE_WIDTH + threadIdx.y)*numBColumns + col];
       }else
       {
           ds_B[ty][tx] = 0;
       }
       __syncthreads();
     
      if((row < numCRows) && (col < numCColumns))
      {
       for(int i = 0; i < TILE_WIDTH; i++)
           Cvalue += ds_A[ty][i] * ds_B[i][tx];
      } 
       __syncthreads();
   }
   if((row < numCRows)&&(col < numCColumns))
   {
     C[row * numCColumns + col] = Cvalue;
   }
}

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  //@@ Set numCRows and numCColumns
  numCRows    = numARows;
  numCColumns = numBColumns;
  //@@ Allocate the hostC matrix
  hostC = (float *)malloc(numCRows * numCColumns * sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Allocate GPU memory here
  int sizeA = numARows * numAColumns * sizeof(float);
  hipMalloc((void **)&deviceA, sizeA);
  int sizeB = numBRows * numBColumns * sizeof(float);
  hipMalloc((void **)&deviceB, sizeB);
  int sizeC = numCRows * numCColumns * sizeof(float);
  hipMalloc((void **)&deviceC, sizeC);

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  //@@ Copy memory to the GPU here
  hipMemcpy(deviceA, hostA, sizeA, hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, sizeB, hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //@@ Initialize the grid and block dimensions here
  dim3 blockDim(16, 16);
  dim3 gridDim(ceil(((float)numCColumns)/blockDim.x), 
               ceil(((float)numCRows)/blockDim.y));

  wbTime_start(Compute, "Performing CUDA computation");
  //@@ Launch the GPU Kernel here
  matrixMultiplyShared<<<gridDim, blockDim>>>(deviceA, deviceB, deviceC,
          numARows,numAColumns,numBRows,numBColumns,numCRows,numCColumns);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, sizeC, hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostC, numCRows, numCColumns);

  free(hostA);
  free(hostB);
  free(hostC);

  return 0;
}
