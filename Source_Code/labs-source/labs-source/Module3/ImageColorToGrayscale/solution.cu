#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ INSERT CODE HERE
#define CHANNELS 3
__global__ void imageColorToGrayscale(float *rgbImageData,
	       	float *grayImageData, int imageChannels, int height, int width){

	int row;
	int col;
	row =  threadIdx.y + blockIdx.y * blockDim.y;
	col =  threadIdx.x + blockIdx.x * blockDim.x;
	
	if(row < height && col < width){
		//get 1D coordinate for the grayscale image
		int grayOffset = row * width + col;
		//one can think of the RGB image having 
		//CHANNLES times colums than the grayscale image
		int rgbOffset = grayOffset * imageChannels;
		float r = rgbImageData[rgbOffset]; //red value for pixel
		float g = rgbImageData[rgbOffset + 1]; //green value for pixel
		float b = rgbImageData[rgbOffset + 2]; // blue value for pixel
		
		//perform the rescaling and store it
		//we multiply by floating point constans
		grayImageData[grayOffset] = 0.21*r + 0.71f*g + 0.07f*b;

	}

}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int imageChannels;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  float *deviceInputImageData;
  float *deviceOutputImageData;

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);

  inputImage = wbImport(inputImageFile);

  imageWidth  = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  // For this lab the value is always 3
  imageChannels = wbImage_getChannels(inputImage);

  // Since the image is monochromatic, it only contains one channel
  outputImage = wbImage_new(imageWidth, imageHeight, 1);

  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  hipMalloc((void **)&deviceInputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float));
  hipMalloc((void **)&deviceOutputImageData,
             imageWidth * imageHeight * sizeof(float));
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  hipMemcpy(deviceInputImageData, hostInputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float),
             hipMemcpyHostToDevice);
  wbTime_stop(Copy, "Copying data to the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ INSERT CODE HERE
  dim3 DimGrid(ceil(imageWidth/16.0), ceil(imageHeight/16.0), 1);
  dim3 DimBlock(16, 64 , 1);

  imageColorToGrayscale<<<DimGrid, DimBlock>>>(deviceInputImageData,
		  deviceOutputImageData, imageChannels, 
                  imageHeight, imageWidth); 
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Doing the computation on the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Copy, "Copying data from the GPU");
  hipMemcpy(hostOutputImageData, deviceOutputImageData,
             imageWidth * imageHeight * sizeof(float),
             hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  wbSolution(args, outputImage);

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
