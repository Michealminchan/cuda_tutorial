#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <wb.h>

int main(int argc, char **argv) {
  wbArg_t args;
  float *hostInput, *hostOutput; // The input 1D list
  int num_elements;              // number of elements in the input list

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput =
      (float *)wbImport(wbArg_getInputFile(args, 0), &num_elements);
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The number of input elements in the input is ",
        num_elements);

  // Declare and allocate the host output array
  //@@ Insert code here
  hostOutput = (float *)malloc(num_elements * sizeof(float));

  // Declare and allocate thrust device input and output vectors
  wbTime_start(GPU, "Allocating GPU memory.");
  //@@ Insert code here
  thrust::device_vector<float> deviceInput(num_elements);
  thrust::device_vector<float> deviceOutput(num_elements);
  thrust::copy(hostInput, hostInput + num_elements, deviceInput.begin());
  wbTime_stop(GPU, "Allocating GPU memory.");

  // Execute vector addition
  wbTime_start(
      Compute,
      "Doing the computation on the GPU and copying data back to host");
  //@@ Insert Code here
  thrust::inclusive_scan(deviceInput.begin(), deviceInput.end(),
                         deviceOutput.begin());
  thrust::copy(deviceOutput.begin(), deviceOutput.end(), hostOutput);

  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbSolution(args, hostOutput, num_elements);

  free(hostInput);
  free(hostOutput);

  return 0;
}
