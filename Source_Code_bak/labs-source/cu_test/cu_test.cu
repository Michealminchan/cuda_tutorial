#include <hip/hip_runtime.h>
#include <iostream>

__host__ void test() {
	  float a = 12.;
	  double b = 3.;
	  auto c = a * b;
	  std::cout << c << std::endl;
}

int main()
{
  test();
  return 0;
}
